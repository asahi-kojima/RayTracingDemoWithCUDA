#include "hip/hip_runtime.h"
﻿#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include "camera.h"
#include "Sphere.h"

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

namespace
{
	constexpr f32 BaseSize = 200;
	constexpr u32 ScreenWidth = static_cast<u32>(16 * BaseSize);
	constexpr u32 ScreenHeight = static_cast<u32>(9 * BaseSize);
	constexpr u32 ScreenPixelNum = ScreenWidth * ScreenHeight;
	__managed__ vec3 renderTarget[ScreenPixelNum];

	constexpr u32 SampleNum = 10;
	constexpr u32 Depth = 30;

	constexpr f32 MAXFLOAT = FLT_MAX;
}


__device__ void prepareObject(Hitable** list, const u32 MaxObjectNum, u32& actualObjectNum)
{
	list[actualObjectNum++] = new Sphere(vec3(0, -1000, 0), 1000, new Metal(vec3(0.3, 0.3, 0.3) * 1.0f, 0.3f));
	list[actualObjectNum++] = new Sphere(vec3(-12, 1, 2),	1.0f, new Metal(vec3(0.5, 0.8, 0.3) * 0.8f, 0.5f));
	list[actualObjectNum++] = new Sphere(vec3(-8, 1, 0),	1.0f, new Metal(vec3(1, 1, 0.2)		* 0.8f, 0.0f));
	list[actualObjectNum++] = new Sphere(vec3(-4, 1, 0),	1.0f, new Metal(vec3(0.5, 0.8, 0.3) * 0.8f, 0.0f));
	list[actualObjectNum++] = new Sphere(vec3(0, 1, 0),		1.0f, new Metal(vec3(1, 1, 1), 0.0f));
	list[actualObjectNum++] = new Sphere(vec3(4, 1, 0),		1.0f, new Metal(vec3(0.5, 0.8, 0.3) * 0.8f, 0.0f));
	list[actualObjectNum++] = new Sphere(vec3(8, 1, 0),		1.0f, new Metal(vec3(0, 1, 0.9)		* 0.8f, 0.1f));
	list[actualObjectNum++] = new Sphere(vec3(12, 1, 2),	1.0f, new Metal(vec3(0, 1, 0.9)		* 1.0f, 0.0f));

	const u32 objectMaxNum = 10;
	hiprandState s;
	hiprand_init(0, objectMaxNum * objectMaxNum * 20, 0, &s);

	 for (s32 a = -objectMaxNum; a < objectMaxNum; a++)
	 {
	 	for (s32 b = -objectMaxNum; b < objectMaxNum; b++)
	 	{
	 		vec3 center(a + 0.9f * hiprand_uniform(&s), 0.2, b + 0.9 * hiprand_uniform(&s));

	 		if ((center - vec3(4, 0.2f, 0)).length() > 0.9f)
	 		{
	 			list[actualObjectNum++] = new Sphere(center, 0.2, new Metal(vec3(hiprand_uniform(&s) * hiprand_uniform(&s), hiprand_uniform(&s) * hiprand_uniform(&s), hiprand_uniform(&s) * hiprand_uniform(&s))));
	 		}
	 	}
	 }
}

__device__ vec3 getColor(ray r, Hitable* world, const s32 depth)
{
	HitRecord rec;
	bool isHit = world->hit(r, 0.001, MAXFLOAT, rec);

	if (isHit)
	{
		ray scattered;
		vec3 attenuation;

		if (depth >= 0 && rec.pMaterial->scatter(r, rec, attenuation, scattered))
		{
			vec3 resultColor = getColor(scattered, world, depth - 1);
			return attenuation * resultColor;
		}
		else
		{
			return vec3(0, 0, 0);
		}
	}
	else
	{
		vec3 unitDirection = normalize(r.direction());
		float t = 0.5f * (unitDirection.y() + 1.0f);
		return vec3(1.0f, 1.0f, 1.0f)* (1.0f - t) + vec3(0.5f, 0.7f, 1.0f) * t;
	}
}

__device__ bool getColorFromRay(ray* pRay, Hitable* world, const s32 depth, vec3* pV)
{
	ray r = *pRay;
	HitRecord rec;
	bool isHit = world->hit(r, 0.001, MAXFLOAT, rec);

	if (isHit)
	{
		ray scattered;
		vec3 attenuation;

		if (depth < Depth && rec.pMaterial->scatter(r, rec, attenuation, scattered))
		{
			*pV = attenuation;
			*pRay = scattered;
			return false;
		}
		else
		{
			*pV = vec3(0, 0, 0);
			return true;
		}
	}
	else
	{
		vec3 unitDirection = normalize(r.direction());
		float t = 0.5f * (unitDirection.y() + 1.0f);
		*pV = vec3(1.0f, 1.0f, 1.0f) * (1.0f - t) + vec3(0.5f, 0.7f, 1.0f) * t;
		return true;
	}
}

__device__ vec3 collectColor(ray r, Hitable* world)
{
	vec3 resultColor(1,1,1);

	ray currentRay = r;

	for (u32 depth = 0; depth < Depth; depth++)
	{
		vec3 colorFromThisRay;
		bool isRayTerminated = getColorFromRay(&currentRay, world, depth, &colorFromThisRay);
		resultColor *= colorFromThisRay;

		if (isRayTerminated)
		{
			break;
		}

	}

	return resultColor;
}

__global__ void castRayToWorld(Camera camera, Hitable* world)
{
	const u32 xid = threadIdx.x + blockDim.x * blockIdx.x;
	const u32 yid = threadIdx.y + blockDim.y * blockIdx.y;
	const u32 index = yid * ScreenWidth + xid;

	if (!(xid < ScreenWidth && yid < ScreenHeight))
	{
		return;
	}

	vec3 color(0,0,0);
	for (u32 sampleNo = 0; sampleNo < SampleNum; sampleNo++)
	{
		f32 x = (static_cast<f32>(xid) + (srandomF64() * 0.5))/ (ScreenWidth - 1);
		f32 y = (static_cast<f32>(yid) + (srandomF64() * 0.5))/ (ScreenHeight - 1);

		ray r = camera.getRay(x, y);
		vec3 resultColor = collectColor(r, world);
		color += resultColor;
	}


	color /= SampleNum;

	renderTarget[index] = color;
}

__global__ void startRayTracing()
{
	constexpr u32 MaxObjectNum = 100;
	u32 actualObjectNum = 0;
	vec3 objectPos[MaxObjectNum];

	Hitable** list = new Hitable * [MaxObjectNum + 1];
	////事前準備
	prepareObject(list, MaxObjectNum, actualObjectNum);

	//カメラの準備
	vec3 lookFrom(13, 2, 5);
	vec3 lookAt(0, 0, 0);
	Camera camera(lookFrom, lookAt, vec3(0, 1, 0), 20, f32(ScreenWidth) / f32(ScreenHeight), 0.01, (lookFrom - lookAt).length());

	Hitable* world = new HitableList(list, actualObjectNum);
	//レイを飛ばす
	dim3 block(16, 16);//スレッドブロック
	dim3 grid((ScreenWidth + block.x - 1) / block.x, (ScreenHeight + block.y - 1) / block.y);
	castRayToWorld << <grid, block >> > (camera, world);
}


int main()
{
	startRayTracing << <1, 1 >> > ();
	CHECK(hipDeviceSynchronize());
	printf("Finish RayTracing\n");
	std::ofstream outputFile("renderResult.ppm");
	outputFile << "P3\n" << ScreenWidth << " " << ScreenHeight << "\n255\n";
	for (s32 yid = ScreenHeight - 1; yid >= 0; yid--)
	{
		for (u32 xid = 0; xid < ScreenWidth; xid++)
		{
			const u32 index = yid * ScreenWidth + xid;
			vec3& col = renderTarget[index];
			col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
			outputFile << int(255.99 * col[0]) << " " << int(255.99 * col[1]) << " " << int(255.99 * col[2]) << "\n";
		}
	}
	outputFile.close();

	return 0;
}
